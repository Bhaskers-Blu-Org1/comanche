/*
 * Orginially From Baidu Research DeepBench.
 * Modifed to test Dawn
 * Author: Yanzhao Wu
 */
#include <boost/program_options.hpp>

#include <chrono>
#include <iomanip>
#include <memory>
#include <stdexcept>
#include <tuple>

#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hiprand.h>

#include <thrust/device_ptr.h>
#include <thrust/fill.h>

#include <cudautility.hpp>

#include "tensor.h"
#include "cudnn_helper.h"
#include "rnn_problems.h"

#include <common/exceptions.h>
#include <api/components.h>
#include <common/dump_utils.h>
#include <common/logging.h>
#include <api/kvstore_itf.h>
#include <config_comanche.h>

#include <api/block_itf.h>
#include "core/physical_memory.h"
#include "core/xms.h"
#include "api/memory_itf.h"

using pool_t = uint64_t;
using namespace Component;

/*
Usage:

The default precision is set based on the architecture and mode.

By default, the program runs the benchmark in training mode.

bin/rnn_bench

To run inference mode, use the following command:

bin/rnn_bench inference


To change the precision for training/inference, use:

bin/rnn_bench train <precision>
bin/rnn_bench inference <precision>

Supported precision types:

For Maxwell GPUS:
float for training and inference

For Pascal GPUS:
float, half for training
float, half, int8 for inference

*/

#ifndef USE_TENSOR_CORES
#if CUDNN_MAJOR >= 7
#define USE_TENSOR_CORES 1
#else
#define USE_TENSOR_CORES 0
#endif
#endif


hipdnnHandle_t cudnn_handle;
hiprandGenerator_t curand_gen;


class cudnnDropout {
    std::shared_ptr<hipdnnDropoutDescriptor_t> dropout_desc_;
    std::shared_ptr<Tensor<uint8_t>> dropout_state_;

    struct DropoutDeleter {
        void operator()(hipdnnDropoutDescriptor_t * dropout_desc) {
            hipdnnDestroyDropoutDescriptor(*dropout_desc);
            delete dropout_desc;
        }
    };

    public:

    cudnnDropout(float dropout_percentage) : dropout_desc_(new hipdnnDropoutDescriptor_t,
                                                           DropoutDeleter()) {
        size_t dropoutStateSize;
        CHECK_CUDNN_ERROR(hipdnnCreateDropoutDescriptor(dropout_desc_.get()));
        CHECK_CUDNN_ERROR(hipdnnDropoutGetStatesSize(cudnn_handle, &dropoutStateSize));

        dropout_state_.reset(new Tensor<uint8_t>(std::vector<int>{static_cast<int>(dropoutStateSize), 1}));

        CHECK_CUDNN_ERROR(hipdnnSetDropoutDescriptor(*dropout_desc_,
                                                    cudnn_handle,
                                                    dropout_percentage,
                                                    dropout_state_->begin(),
                                                    dropoutStateSize,
                                                    0ULL) );
    }

    hipdnnDropoutDescriptor_t desc() const { return *dropout_desc_; }
};

template <typename T>
class cudnnRNN {
    RNNDescriptor<T> rnn_desc_;
    FilterDescriptorNd<T> wDesc_;
    cudnnDropout dropout_;

    int time_steps_;

    TensorDescriptorNdArray<T> xDescArray_;
    TensorDescriptorNdArray<T> yDescArray_;
    TensorDescriptorNdArray<T> dxDescArray_;
    TensorDescriptorNdArray<T> dyDescArray_;

    TensorDescriptorNd<T> hx_desc_;
    TensorDescriptorNd<T> hy_desc_;
    TensorDescriptorNd<T> dhx_desc_;
    TensorDescriptorNd<T> dhy_desc_;
    TensorDescriptorNd<T> cx_desc_;
    TensorDescriptorNd<T> cy_desc_;
    TensorDescriptorNd<T> dcx_desc_;
    TensorDescriptorNd<T> dcy_desc_;

    size_t weight_size_;
    size_t workspace_size_;
    size_t train_size_;

    Tensor<T> weights_;
    Tensor<T> dW_;

    Tensor<float> workspace_;
    Tensor<float> trainspace_;

    public:

    cudnnRNN(int hidden_size, int batch_size, int time_steps, const std::string& rnn_type) :
        dropout_(0.f), time_steps_(time_steps),
        xDescArray_({batch_size, hidden_size, 1}, {hidden_size, 1, 1}, time_steps),
        yDescArray_({batch_size, hidden_size, 1}, {hidden_size, 1, 1}, time_steps),
        dxDescArray_({batch_size, hidden_size, 1}, {hidden_size, 1, 1}, time_steps),
        dyDescArray_({batch_size, hidden_size, 1}, {hidden_size, 1, 1}, time_steps),
        hx_desc_({1, batch_size, hidden_size}, {hidden_size * batch_size, hidden_size, 1}),
        hy_desc_({1, batch_size, hidden_size}, {hidden_size * batch_size, hidden_size, 1}),
        dhx_desc_({1, batch_size, hidden_size}, {hidden_size * batch_size, hidden_size, 1}),
        dhy_desc_({1, batch_size, hidden_size}, {hidden_size * batch_size, hidden_size, 1}),
        cx_desc_({1, batch_size, hidden_size}, {hidden_size * batch_size, hidden_size, 1}),
        cy_desc_({1, batch_size, hidden_size}, {hidden_size * batch_size, hidden_size, 1}),
        dcx_desc_({1, batch_size, hidden_size}, {hidden_size * batch_size, hidden_size, 1}),
        dcy_desc_({1, batch_size, hidden_size}, {hidden_size * batch_size, hidden_size, 1})
        {


            rnn_desc_ = RNNDescriptor<T>(hidden_size,
                                             1,
                                             dropout_.desc(),
                                             HIPDNN_SKIP_INPUT,
                                             HIPDNN_UNIDIRECTIONAL,
                                             rnn_type,
                                             cudnn_handle);
            hipdnnDataType_t type;
            if (std::is_same<T, float>::value)
                type = HIPDNN_DATA_FLOAT;
#if CUDNN_MAJOR >= 6
            else if (std::is_same<T, uint8_t>::value)
                type = HIPDNN_DATA_INT8;
#endif
            else if (std::is_same<T, uint16_t>::value)
                type= HIPDNN_DATA_HALF;
            else 
                throw std::runtime_error("Unknown type in cudnnRNN constructor.");

            CHECK_CUDNN_ERROR( hipdnnGetRNNParamsSize(cudnn_handle,
                                                     rnn_desc_.desc(),
                                                     xDescArray_.ptr()[0],
                                                     &weight_size_,
                                                     type) );

#if (CUDNN_MAJOR >= 7) && (USE_TENSOR_CORES)
            CHECK_CUDNN_ERROR( cudnnSetRNNMatrixMathType(rnn_desc_.desc(), HIPDNN_TENSOR_OP_MATH) );
#endif

            weights_ = rand<T>(std::vector<int>{static_cast<int>(weight_size_ / sizeof(T)), 1}, curand_gen);

            std::vector<int> dim = {weights_.size(), 1, 1};
            wDesc_ = FilterDescriptorNd<T>(HIPDNN_TENSOR_NCHW, dim);

            CHECK_CUDNN_ERROR( hipdnnGetRNNWorkspaceSize(cudnn_handle,
                                                        rnn_desc_.desc(),
                                                        time_steps,
                                                        xDescArray_.ptr(),
                                                        &workspace_size_) );

            dW_ = zeros<T>(std::vector<int>{static_cast<int>(weight_size_ / sizeof(T)), 1});

            workspace_ = zeros<float>(std::vector<int>{static_cast<int>(workspace_size_ / sizeof(float)), 1});

            CHECK_CUDNN_ERROR( hipdnnGetRNNTrainingReserveSize(cudnn_handle,
                                                              rnn_desc_.desc(),
                                                              time_steps,
                                                              xDescArray_.ptr(),
                                                              &train_size_) );
            trainspace_ = zeros<float>(std::vector<int>{static_cast<int>(train_size_ / sizeof(float)), 1});
        }
        void forward(Tensor<T> x, Tensor<T> hx, Tensor<T> cx,
                     Tensor<T> y, Tensor<T> hy, Tensor<T> cy) {
            CHECK_CUDNN_ERROR( hipdnnRNNForwardTraining(cudnn_handle,
                                                       rnn_desc_.desc(),
                                                       time_steps_,
                                                       xDescArray_.ptr(),
                                                       (void *)x.begin(),
                                                       hx_desc_.desc(),
                                                       (void *)hx.begin(),
                                                       cx_desc_.desc(),
                                                       (void *)cx.begin(),
                                                       wDesc_.desc(),
                                                       (void *)weights_.begin(),
                                                       yDescArray_.ptr(),
                                                       (void *)y.begin(),
                                                       hy_desc_.desc(),
                                                       (void *)hy.begin(),
                                                       cy_desc_.desc(),
                                                       (void *)cy.begin(),
                                                       (void *)workspace_.begin(),
                                                       workspace_size_,
                                                       (void *)trainspace_.begin(),
                                                       train_size_) );
        }
        void backward_data(Tensor<T> y, Tensor<T> dy, Tensor<T> dhy,
                           Tensor<T> dcy, Tensor<T> hx, Tensor<T> cx,
                           Tensor<T> dx, Tensor<T> dhx, Tensor<T> dcx) {
            CHECK_CUDNN_ERROR( hipdnnRNNBackwardData(cudnn_handle,
                                                    rnn_desc_.desc(),
                                                    time_steps_,
                                                    yDescArray_.ptr(),
                                                    (void *)y.begin(),
                                                    dyDescArray_.ptr(),
                                                    (void *)dy.begin(),
                                                    dhy_desc_.desc(),
                                                    (void *)dhy.begin(),
                                                    dcy_desc_.desc(),
                                                    (void *)dcy.begin(),
                                                    wDesc_.desc(),
                                                    (void *)weights_.begin(),
                                                    hx_desc_.desc(),
                                                    (void *)hx.begin(),
                                                    cx_desc_.desc(),
                                                    (void *)cx.begin(),
                                                    dxDescArray_.ptr(),
                                                    (void *)dx.begin(),
                                                    dhx_desc_.desc(),
                                                    (void *)dhx.begin(),
                                                    dcx_desc_.desc(),
                                                    (void *)dcx.begin(),
                                                    (void *)workspace_.begin(),
                                                    workspace_size_,
                                                    (void *)trainspace_.begin(),
                                                    train_size_) );
        }

        void backward_params(Tensor<T> x, Tensor<T> hx, Tensor<T> y) {
            CHECK_CUDNN_ERROR(hipdnnRNNBackwardWeights(cudnn_handle,
                                                      rnn_desc_.desc(),
                                                      time_steps_,
                                                      xDescArray_.ptr(),
                                                      (void *)x.begin(),
                                                      hx_desc_.desc(),
                                                      (void *)hx.begin(),
                                                      yDescArray_.ptr(),
                                                      (void *)y.begin(),
                                                      (void *)workspace_.begin(),
                                                      workspace_size_,
                                                      wDesc_.desc(),
                                                      (void *)dW_.begin(),
                                                      (void *)trainspace_.begin(),
                                                      train_size_) );
        }

};

template <typename T>
std::tuple<int, int, int> time_rnn(int hidden_size,
                                   int batch_size,
                                   int time_steps,
                                   const std::string& type,
                                   int inference,
								   Component::IKVStore * store,
								   pool_t pool) {

    cudnnRNN<T> rnn(hidden_size, batch_size, time_steps, type);

    auto x  = rand<T>({hidden_size, batch_size * time_steps}, curand_gen);
    std::string key_name("Task");
    key_name = key_name+"_hs" + std::to_string(hidden_size)\
    		+ "_bs" + std::to_string(batch_size)\
			+ "_ts" + std::to_string(time_steps);
    auto input = zeros<T>({hidden_size, batch_size * time_steps});
    auto handle = store -> register_direct_memory((void*) input.begin(), input.size());
    assert(handle);

    //write_to_file(key_name, x);
    put_into_kvstore(key_name, x, store, pool);

    auto y  = rand<T>({hidden_size, batch_size * time_steps}, curand_gen);
    auto dx = rand<T>({hidden_size, batch_size * time_steps}, curand_gen);
    auto dy = rand<T>({hidden_size, batch_size * time_steps}, curand_gen);

    auto hx = rand<T>({hidden_size, batch_size}, curand_gen);
    auto hy = rand<T>({hidden_size, batch_size}, curand_gen);
    auto cx = rand<T>({hidden_size, batch_size}, curand_gen);
    auto cy = rand<T>({hidden_size, batch_size}, curand_gen);
    auto dhx = rand<T>({hidden_size, batch_size}, curand_gen);
    auto dhy = rand<T>({hidden_size, batch_size}, curand_gen);
    auto dcx = rand<T>({hidden_size, batch_size}, curand_gen);
    auto dcy = rand<T>({hidden_size, batch_size}, curand_gen);

    int numRepeats = 100;

    //Warm up
    size_t input_size = input.size();
    //read_from_file(key_name, input);
    status_t rc = store->get_direct(pool, key_name, input.begin(), input_size, handle);
    rnn.forward(input, hx, cx, y, hy, cy);

    hipDeviceSynchronize();

    auto start = std::chrono::steady_clock::now();

    for (int i = 0; i < numRepeats; ++i) {
        //read_from_file(key_name, input);
        rc = store->get_direct(pool, key_name, input.begin(), input_size, handle);
    	assert(rc == S_OK);
        //rnn.forward(input, hx, cx, y, hy, cy);
    }
    hipDeviceSynchronize();

    auto end = std::chrono::steady_clock::now();
    auto micro_sec = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
    PLOG("Read from remote kvstore takes %lf us", micro_sec/(double)numRepeats);


    auto forward_time = std::chrono::duration<double, std::micro>(end - start).count() / numRepeats;
    int bwd_data_time = 0;
    int bwd_params_time = 0;

    if (!inference) {
        //Warm up
        rnn.backward_data(y, dy, dhy, dcy,
                          hx, cx, dx, dhx, dcx);

        hipDeviceSynchronize();

        start = std::chrono::steady_clock::now();

        for (int i = 0; i < numRepeats; ++i) {
            rnn.backward_data(y, dy, dhy, dcy,
                              hx, cx, dx, dhx, dcx);
        }
        hipDeviceSynchronize();

        end = std::chrono::steady_clock::now();
        bwd_data_time = std::chrono::duration<double, std::micro>(end - start).count() / numRepeats;

        /* Backward wrt params */
        //Warm up
        rnn.backward_params(x, hx, y);

        hipDeviceSynchronize();

        start = std::chrono::steady_clock::now();

        for (int i = 0; i < numRepeats; ++i) {
            rnn.backward_params(x, hx, y);
        }

        hipDeviceSynchronize();

        end = std::chrono::steady_clock::now();
        bwd_params_time = std::chrono::duration<double, std::micro>(end - start).count() / numRepeats;


    }

    return std::make_tuple(static_cast<int>(forward_time),
                           static_cast<int>(bwd_data_time),
                           static_cast<int>(bwd_params_time));

}

Component::IKVStore * create_store(const std::string& addr,
                                   const std::string& device,
                                   const unsigned debug_level) {
  using namespace Component;

  std::string path = CONF_COMANCHE_INSTALL;
  path += "/lib/libcomanche-dawn-client.so";

  IBase * comp = load_component(path.c_str(), dawn_client_factory);
  assert(comp);
  IKVStore_factory * fact = (IKVStore_factory *) comp->query_interface(IKVStore_factory::iid());

  IKVStore * inst = nullptr;

  inst = fact->create(debug_level,
                      "rnnbench",
                      addr.c_str(),
                      device.c_str());

  fact->release_ref();
  return inst;
}


int main(int argc, char **argv) {

    int inference = 0;

    namespace po = boost::program_options;
       po::options_description desc("Options");
       desc.add_options()
       		("dawn-server", po::value<std::string>()->default_value("10.0.0.22:11911"))
   			("debug", po::value<unsigned>()->default_value(0))
   			("device", po::value<std::string>()->default_value("mlx5_0"))
   			("inference", po::value<int>()->default_value(0))
   			("precision", po::value<std::string>()->default_value("float"))
   			("help", "Show this help")
   			;
       po::variables_map vm;
       po::store(po::parse_command_line(argc, argv, desc), vm);

       if(vm.count("help")) {
       	std::cout << desc;
       	return -1;
       }
       inference = vm["inference"].as<int>();

#if CUDNN_MAJOR >= 6
    std::string precision;
    if (inference)
        precision = "int8";
    else
        precision = "half";
#else
    std::string precision = "float";
#endif

    std::cout << vm.count("precision") << vm["precision"].as<std::string>() << std::endl;
	if (vm.count("precision")) {
		precision = vm["precision"].as<std::string>();
	}

	// Create KVStore
	Component::IKVStore *  store = create_store(vm["dawn-server"].as<std::string>(),
								vm["device"].as<std::string>(),
								vm["debug"].as<unsigned>());
	// Create pool
	auto pool = store->create_pool("/poolsgpu0", MB(128));

	hipFree(0);
	CHECK_CUDNN_ERROR( hipdnnCreate(&cudnn_handle) );

	hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(curand_gen, 123ULL);

    if (inference) {
        std::cout << std::setw(45) << "Running inference benchmark " << std::endl;
    } else {
        std::cout << std::setw(45) << "Running training benchmark " << std::endl;
    }

    std::cout << std::setw(30) << "Times" << std::endl;
    std::cout << std::setfill('-') << std::setw(115) << "-" << std::endl;
    std::cout << std::setfill(' ');
    std::cout << "    type    hidden   N     timesteps   precision   fwd_time (usec)   ";
    if (!inference) {
        std::cout << "bwd_inputs_time (usec)";
        std::cout << "  bwd_params_time (usec)";
    }

    std::cout << std::endl;
    for (const auto &problem : (inference ? inference_server_set : training_set)) {
        int hidden_state, batch_size, time_steps;
        std::string type;
        std::tie(hidden_state, batch_size, time_steps, type) = problem;

        std::cout << std::setw(8) << type;
        std::cout << std::setw(8) << hidden_state;
        std::cout << std::setw(8) << batch_size;
        std::cout << std::setw(8) << time_steps;
        std::cout << std::setw(14) << precision;
        int fwd_time, bwd_data_time, bwd_params_time;

        std::stringstream ss;
        ss << "Unsupported precision requested. Precision: " << precision << " Inference: " << inference;

#if CUDNN_MAJOR >= 6
        if (inference) {
            if (precision == "float") {
                std::tie(fwd_time, bwd_data_time, bwd_params_time) =
                    time_rnn<float>(hidden_state,
                                    batch_size,
                                    time_steps,
                                    type,
                                    inference, store, pool);

            } else if (precision == "half") {
                std::tie(fwd_time, bwd_data_time, bwd_params_time) =
                    time_rnn<uint16_t>(hidden_state,
                                       batch_size,
                                       time_steps,
                                       type,
                                       inference, store, pool);
            } else if (precision == "int8") {
                std::tie(fwd_time, bwd_data_time, bwd_params_time) =
                    time_rnn<uint8_t>(hidden_state,
                                      batch_size,
                                      time_steps,
                                      type,
                                      inference, store, pool);
            } else {
                throw std::runtime_error(ss.str());
            }
        } else {
            if (precision == "float") {
                std::tie(fwd_time, bwd_data_time, bwd_params_time) =
                     time_rnn<float>(hidden_state,
                                     batch_size,
                                     time_steps,
                                     type,
                                     inference, store, pool);

            } else if (precision == "half") {
                std::tie(fwd_time, bwd_data_time, bwd_params_time) =
                     time_rnn<uint16_t>(hidden_state,
                                        batch_size,
                                        time_steps,
                                        type,
                                        inference, store, pool);
            } else {
                throw std::runtime_error(ss.str());
            }
        }
#else
        if (precision != "float")
            throw std::runtime_error(ss.str());

        std::tie(fwd_time, bwd_data_time, bwd_params_time) =
             time_rnn<float>(hidden_state,
                             batch_size,
                             time_steps,
                             type,
                             inference, store, pool);
#endif

        std::cout << std::setw(18) << fwd_time;
        if (!inference) {
            std::cout << std::setw(20) << bwd_data_time;
            std::cout << std::setw(20) << bwd_params_time;
        }
        std::cout << std::endl;
    }

    // Close the pool
	store->close_pool(pool);
	store->delete_pool("/poolsgpu0");
	// Release the KVStore
	store->release_ref();


    hipdnnDestroy(cudnn_handle);
    hiprandDestroyGenerator(curand_gen);

    return 0;
}
