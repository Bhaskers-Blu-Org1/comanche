/*
 * Orginially From Baidu Research DeepBench.
 * Modifed to test Dawn
 * Author: Yanzhao Wu
 */
#include <boost/program_options.hpp>

#include <iomanip>
#include <memory>
#include <chrono>
#include <vector>
#include <tuple>
#include <chrono>

#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <hiprand.h>

#include <thrust/device_ptr.h>
#include <thrust/fill.h>

#include <cudautility.hpp>

#include "tensor.h"
#include "cudnn_helper.h"
#include "conv_problems.h"

#include <common/exceptions.h>
#include <api/components.h>
#include <common/dump_utils.h>
#include <common/logging.h>
#include <api/kvstore_itf.h>
#include <config_comanche.h>

#include <api/block_itf.h>
#include "core/physical_memory.h"
#include "core/xms.h"
#include "api/memory_itf.h"

#define USE_GET 0

#ifndef PAD_KERNELS
#define PAD_KERNELS 1
#endif

#ifndef USE_TENSOR_CORES
#if CUDNN_MAJOR >= 7
#define USE_TENSOR_CORES 1
#else
#define USE_TENSOR_CORES 0
#endif
#endif

using pool_t = uint64_t;
using namespace Component;
/*
Usage:

The default precision is set based on the architecture and mode.

By default, the program runs the benchmark in training mode.

./conv_bench

To run inference mode, use the following command:

./conv_bench inference


To change the precision for training/inference, use:

./conv_bench train <precision>
./conv_bench inference <precision>

Supported precision types:

For Maxwell GPUS: 
float for training and inference

For Pascal GPUS:
float, half for training
float, half, int8 for inference

*/

// T1 is used as the data type for inputs, weights and outputs. 
// T2 is used to describe the compute precision. This is used in inference mode in the INT8_CONFIG
template <typename T1, typename T2>
class cudnnCNN {
    TensorDescriptor4d<T1> x_desc_;
    TensorDescriptor4d<T1> h_desc_;

    FilterDescriptor4d<T1> w_desc_;

    std::vector<int> output_dims_;
    int num_repeats_;

    size_t fwd_workspace_size_;
    size_t bwd_inputs_workspace_size_;
    size_t bwd_params_workspace_size_;

    Tensor<float> fwd_workspace_;
    Tensor<float> bwd_inputs_workspace_;
    Tensor<float> bwd_params_workspace_;

    hipdnnConvolutionFwdAlgo_t fwd_algo_;
    hipdnnConvolutionBwdDataAlgo_t bwd_inputs_algo_;
    hipdnnConvolutionBwdFilterAlgo_t bwd_params_algo_;

    const float alpha_ = 1.f;
    const float beta_  = 0.f;

    ConvolutionDescriptor<T2> conv_desc_;
    CudnnHandle cudnn_handle_;

public:

    cudnnCNN(int w, int h, int c, int n, int k, int r, int s,
             int pad_w, int pad_h, int wstride, int hstride,
             int inference)
             :
        cudnn_handle_(),
        conv_desc_(pad_h, pad_w, hstride, wstride)
    {
        int out_h, out_w, out_c, out_n;

        hipdnnTensorFormat_t format;
        // For int8 inference, the supported format is NHWC
        if (std::is_same<T1, uint8_t>::value) {
            format = HIPDNN_TENSOR_NHWC;
        } else {
            format = HIPDNN_TENSOR_NCHW;
        }

        x_desc_ = TensorDescriptor4d<T1>(format, n, c, h, w);
        w_desc_ = FilterDescriptor4d<T1>(format, k, c, r, s);

#if (CUDNN_MAJOR >= 7) && (USE_TENSOR_CORES)
        hipdnnSetConvolutionMathType(conv_desc_.desc(), HIPDNN_TENSOR_OP_MATH);
#endif
        // Get output dimensions
        CHECK_CUDNN_ERROR(hipdnnGetConvolution2dForwardOutputDim(conv_desc_.desc(),
                                                                x_desc_.desc(),
                                                                w_desc_.desc(),
                                                                &out_n,
                                                                &out_c,
                                                                &out_h,
                                                                &out_w));

        h_desc_ = TensorDescriptor4d<T1>(format, out_n, out_c, out_h, out_w);

        output_dims_ = {out_w, out_h, out_c, out_n};

#if USE_GET
        if (std::is_same<T1, uint8_t>::value) {
            //Note: cuDNN only supports IMPLICIT_PRECOMP_GEMM for int8 data type.
            fwd_algo_ = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
        } else {
        // Pick forward convolution algorithm
        CHECK_CUDNN_ERROR(hipdnnGetConvolutionForwardAlgorithm(cudnn_handle_.handle(),
                                                              x_desc_.desc(),
                                                              w_desc_.desc(),
                                                              conv_desc_.desc(),
                                                              h_desc_.desc(),
                                                              HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                              0,
                                                              &fwd_algo_));
        }
#else
       // Pick forward convolution algorithm
        hipdnnConvolutionFwdAlgoPerf_t fwd_perf;
        int ret_count;

        if (std::is_same<T1, uint8_t>::value) {
            //Note: cuDNN only supports IMPLICIT_PRECOMP_GEMM for int8 data type.
            fwd_algo_ = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
        } else {
            CHECK_CUDNN_ERROR(hipdnnFindConvolutionForwardAlgorithm(cudnn_handle_.handle(),
                                                                   x_desc_.desc(),
                                                                   w_desc_.desc(),
                                                                   conv_desc_.desc(),
                                                                   h_desc_.desc(),
                                                                   1,
                                                                   &ret_count,
                                                                   &fwd_perf));
            fwd_algo_ = fwd_perf.algo;
        }
#endif
#if (CUDNN_MAJOR >= 7) && (USE_TENSOR_CORES)
        // Tensor Op math only supports IMPLICIT_PRECOMP_GEMM algorithm
        fwd_algo_ = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
#endif
        if (std::is_same<T1, uint8_t>::value) {
            //Note: cudnn workspace size function doesn't work for INT8_CONFIG
            fwd_workspace_size_= 1073741824;
        } else {
            // Set fwd workspace size
            CHECK_CUDNN_ERROR(hipdnnGetConvolutionForwardWorkspaceSize(cudnn_handle_.handle(),
                                                                      x_desc_.desc(),
                                                                      w_desc_.desc(),
                                                                      conv_desc_.desc(),
                                                                      h_desc_.desc(),
                                                                      fwd_algo_,
                                                                      &fwd_workspace_size_));
        }

        fwd_workspace_ = zeros<float>(std::vector<int>{static_cast<int>(fwd_workspace_size_ / sizeof(float)), 1});

        if (!inference) {
#if USE_GET
            // Pick backward convolution algorithm
            CHECK_CUDNN_ERROR(hipdnnGetConvolutionBackwardFilterAlgorithm(cudnn_handle_.handle(),
                                                                         x_desc_.desc(),
                                                                         h_desc_.desc(),
                                                                         conv_desc_.desc(),
                                                                         w_desc_.desc(),
                                                                         HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST,
                                                                         0,
                                                                         &bwd_params_algo_));
#else
            hipdnnConvolutionBwdFilterAlgoPerf_t filter_perf;

            if (std::is_same<T1, uint8_t>::value) {

                fwd_algo_ = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;

            }
            CHECK_CUDNN_ERROR(hipdnnFindConvolutionBackwardFilterAlgorithm(cudnn_handle_.handle(),
                                                                         x_desc_.desc(),
                                                                         h_desc_.desc(),
                                                                         conv_desc_.desc(),
                                                                         w_desc_.desc(),
                                                                         1,
                                                                         &ret_count,
                                                                         &filter_perf));
            bwd_params_algo_ = filter_perf.algo;
#endif
#if (CUDNN_MAJOR >= 7) && (USE_TENSOR_CORES)
            // Tensor Op math only supports this algorithm.
            bwd_params_algo_ = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1;
#endif

            // Backward params workspace
            CHECK_CUDNN_ERROR(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cudnn_handle_.handle(),
                                                                             x_desc_.desc(),
                                                                             h_desc_.desc(),
                                                                             conv_desc_.desc(),
                                                                             w_desc_.desc(),
                                                                             bwd_params_algo_,
                                                                             &bwd_params_workspace_size_));



            bwd_params_workspace_ = zeros<float>(std::vector<int>{static_cast<int>(bwd_params_workspace_size_ / sizeof(float)), 1});

#if USE_GET
            // Pick backward wrt inputs convolution algorithm
            CHECK_CUDNN_ERROR(hipdnnGetConvolutionBackwardDataAlgorithm(cudnn_handle_.handle(),
                                                                       w_desc_.desc(),
                                                                       h_desc_.desc(),
                                                                       conv_desc_.desc(),
                                                                       x_desc_.desc(),
                                                                       HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST,
                                                                       0,
                                                                       &bwd_inputs_algo_));
#else
            hipdnnConvolutionBwdDataAlgoPerf_t data_perf;
            CHECK_CUDNN_ERROR(hipdnnFindConvolutionBackwardDataAlgorithm(cudnn_handle_.handle(),
                                                                        w_desc_.desc(),
                                                                        h_desc_.desc(),
                                                                        conv_desc_.desc(),
                                                                        x_desc_.desc(),
                                                                        1,
                                                                        &ret_count,
                                                                        &data_perf));
            bwd_inputs_algo_ = data_perf.algo;
#endif
#if (CUDNN_MAJOR >= 7) && (USE_TENSOR_CORES)
            //Tensor Op math only supports this algorithm.
            bwd_inputs_algo_ = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1;
#endif

            CHECK_CUDNN_ERROR(hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnn_handle_.handle(),
                                                                           w_desc_.desc(),
                                                                           h_desc_.desc(),
                                                                           conv_desc_.desc(),
                                                                           x_desc_.desc(),
                                                                           bwd_inputs_algo_,
                                                                           &bwd_inputs_workspace_size_));

            bwd_inputs_workspace_ = zeros<float>(std::vector<int>{static_cast<int>(bwd_inputs_workspace_size_ / sizeof(float)), 1});
        }

    }

    std::vector<int> get_output_dims() { return output_dims_; }

    std::string get_fwd_algo_string() {
        if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM)
            return "IMPLICIT_GEMM";
        else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM)
            return "IMPLICIT_PRECOMP_GEMM";
        else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_GEMM) 
            return "GEMM";
        else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT)
            return "DIRECT";
        else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_FFT)
            return "FFT";
        else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING)
            return "FFT_TILING";
        else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD)
            return "WINOGRAD";
#if CUDNN_MAJOR >= 6
        else if (fwd_algo_ == HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED)
            return "WINOGRAD_NONFUSED";
#endif
        else {
            std::stringstream ss;
            ss << "Illegal algorithm passed to get_fwd_algo_string. Algo: " << fwd_algo_ << std::endl;
            throw std::runtime_error(ss.str());
        }
    }


    void forward(Tensor<T1> x, Tensor<T1> filter, Tensor<T1> h) {

        // Convolution forward.
        CHECK_CUDNN_ERROR(hipdnnConvolutionForward(cudnn_handle_.handle(),
                                                  &alpha_,
                                                  x_desc_.desc(),
                                                  x.begin(),
                                                  w_desc_.desc(),
                                                  filter.begin(),
                                                  conv_desc_.desc(),
                                                  fwd_algo_,
                                                  fwd_workspace_.begin(),
                                                  fwd_workspace_size_,
                                                  &beta_,
                                                  h_desc_.desc(),
                                                  h.begin()));

    }

    void backward_params(Tensor<T1> x, Tensor<T1> delta, Tensor<T1> dW) {

        CHECK_CUDNN_ERROR(hipdnnConvolutionBackwardFilter(cudnn_handle_.handle(),
                                                         &alpha_,
                                                         x_desc_.desc(),
                                                         x.begin(),
                                                         h_desc_.desc(),
                                                         delta.begin(),
                                                         conv_desc_.desc(),
                                                         bwd_params_algo_,
                                                         bwd_params_workspace_.begin(),
                                                         bwd_params_workspace_size_,
                                                         &beta_,
                                                         w_desc_.desc(),
                                                         dW.begin()));


    }

    void backward_inputs(Tensor<T1> filter, Tensor<T1> delta, Tensor<T1> dX) {

        CHECK_CUDNN_ERROR(hipdnnConvolutionBackwardData(cudnn_handle_.handle(),
                                                      &alpha_,
                                                      w_desc_.desc(),
                                                      filter.begin(),
                                                      h_desc_.desc(),
                                                      delta.begin(),
                                                      conv_desc_.desc(),
                                                      bwd_inputs_algo_,
                                                      bwd_inputs_workspace_.begin(),
                                                      bwd_inputs_workspace_size_,
                                                      &beta_,
                                                      x_desc_.desc(),
                                                      dX.begin()));

    }
};
template <typename T1, typename T2>
std::tuple<int, int, int, std::string> time_cnn(
         int k, int c, int r, int s,
         int n, int h, int w,
         int pad_h, int pad_w,
         int hstride, int wstride,
         int num_repeats,
         hiprandGenerator_t curand_gen,
         int inference,
	 Component::IKVStore * store,
	 pool_t pool
        ) {

    cudnnCNN<T1, T2> cnn(w, h, c, n, k, r, s, pad_w, pad_h, wstride, hstride, inference);

    // Allocate memory for filter
    auto filter = rand<T1>(std::vector<int>{s, r, c, k}, curand_gen);

    auto random_input = rand<T1>(std::vector<int>{w, h, c, n}, curand_gen);
    std::string key_name("Task");
    key_name = key_name+"_k"+std::to_string(k)\
			+"_c"+std::to_string(c)\
			+"_r"+std::to_string(r)\
			+"_s"+std::to_string(s)\
			+"_n"+std::to_string(n)\
			+"_h"+std::to_string(h)\
			+"_w"+std::to_string(w)\
			+"_padH"+std::to_string(pad_h)\
			+"_padW"+std::to_string(pad_w)\
			+"_hstride"+std::to_string(hstride)\
			+"_wstride"+std::to_string(wstride);


    // Allocate memory for input
    auto input = zeros<T1>(std::vector<int>{w, h, c, n});
    // Allocate memory for output tensor
    auto output = zeros<T1>(cnn.get_output_dims());

    //std::string fwd_algo_s = cnn.get_fwd_algo_string();
    //write_to_file(key_name, random_input);
    put_into_kvstore(key_name, random_input, store, pool);

    auto handle = store->register_direct_memory((void*) input.begin(), input.size());
    assert(handle);

    std::string fwd_algo_s = cnn.get_fwd_algo_string();

    //Warm up
    //read_from_file(key_name, input);
    size_t input_size = input.size();
    status_t rc = store->get_direct(pool, key_name, input.begin(), input_size, handle);
    if(rc != S_OK) {
    	PLOG("Error: Get_Direct %d", rc);
    	PLOG("Error Code: S_MORE %d", IKVStore::S_MORE);
    	PLOG("Error Code: E_POOL_NOT_FOUND %d", IKVStore::E_POOL_NOT_FOUND);
    	PLOG("Error Code: E_KEY_NOT_FOUND %d", IKVStore::E_KEY_NOT_FOUND);
    }

    std::chrono::system_clock::time_point read_start = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < num_repeats; ++i) {
        //read_from_file(key_name, input);
        rc=store->get_direct(pool, key_name, input.begin(), input_size, handle);
        assert(rc == S_OK || rc == IKVStore::S_MORE);
    }
    //read_from_file(key_name, input);
    //store->get_direct(file_name, gdr_ptr._h_ptr, input.size());
    std::chrono::system_clock::time_point read_end = std::chrono::high_resolution_clock::now();
    auto micro_sec = std::chrono::duration_cast<std::chrono::microseconds>(read_end - read_start).count();
    PLOG("Read from remote kvstore takes %lf microsecond", micro_sec/(double)num_repeats);
    //void * tmp_space = (void *) malloc(input.size());
    //memset(tmp_space, 0xb, input.size());
    //size_t tmp_space_len = 0;
    //assert(tmp_space);
    //PLOG("GPU_direct read to CPU memory");
    //kvstore->get_direct(file_name, tmp_space, input.size());
    //kvstore->get_direct(file_name, gdr_ptr._h_ptr, input.size());
    //assert(tmp_space_len == input.size());
    //hexdump(tmp_space, 32);
    //kvstore->get_direct(file_name, gdr_ptr._h_ptr, input.size());
    //PLOG("Dump input tensor");
    //hexdump_tensor(input);
    
    //cnn.forward(input, filter, output);
    store->unregister_direct_memory(handle);
    return std::tuple<int, int, int, std::string>(0, 0, 0, fwd_algo_s);
    hipDeviceSynchronize();
    auto start = std::chrono::steady_clock::now();
    /*
    for (int i = 0; i < num_repeats; ++i) {
    	//read_from_file(file_name, input);
    	kvstore->get_direct(file_name, gdr_ptr._h_ptr, input.size());
        cnn.forward(input, filter, output);
    }
    */

    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    int fwd_time = static_cast<int>(std::chrono::duration<double, std::micro>(end - start).count() / num_repeats);

    int bwd_inputs_time = 0;
    int bwd_params_time = 0;

    if (!inference) {
        // Allocate memory for backward pass wrt weights
        auto delta = rand<T1>(cnn.get_output_dims(), curand_gen);
        auto dW = zeros<T1>(std::vector<int>{s, r, c, k});

        // Warm up backward
        cnn.backward_params(input, delta, dW);

        hipDeviceSynchronize();
        start = std::chrono::steady_clock::now();

        for (int i = 0; i < num_repeats; ++i) {
            // Backward pass wrt weights
            cnn.backward_params(input, delta, dW);
        }

        hipDeviceSynchronize();
        end = std::chrono::steady_clock::now();

        bwd_params_time = static_cast<int>(std::chrono::duration<double, std::micro>(end - start).count() / num_repeats);

        //Allocate memory for backward pass wrt inputs
        auto dX = zeros<T1>(std::vector<int>{w, h, c, n});

        //Warm up backward inputs
        cnn.backward_inputs(filter, delta, dX);

        hipDeviceSynchronize();
        start = std::chrono::steady_clock::now();

        for (int i = 0; i < num_repeats; ++i) {
            // Backward pass wrt weights
            cnn.backward_inputs(filter, delta, dX);

        }

        hipDeviceSynchronize();
        end = std::chrono::steady_clock::now();

        bwd_inputs_time = static_cast<int>(std::chrono::duration<double, std::micro>(end - start).count() / num_repeats);
    }

    return std::tuple<int, int, int, std::string>(fwd_time, bwd_inputs_time, bwd_params_time, fwd_algo_s);

}

Component::IKVStore * create_store(const std::string& addr,
                                   const std::string& device,
                                   const unsigned debug_level) {
  using namespace Component;

  std::string path = CONF_COMANCHE_INSTALL;
  path += "/lib/libcomanche-dawn-client.so";

  IBase * comp = load_component(path.c_str(), dawn_client_factory);
  assert(comp);
  IKVStore_factory * fact = (IKVStore_factory *) comp->query_interface(IKVStore_factory::iid());

  IKVStore * inst = nullptr;

  inst = fact->create(debug_level,
                      "convbench",
                      addr.c_str(),
                      device.c_str());

  fact->release_ref();
  return inst;
}

int main(int argc, char **argv) {

    int num_repeats = 300;

    int inference = 0;

    namespace po = boost::program_options;
    po::options_description desc("Options");
    desc.add_options()
    		("dawn-server", po::value<std::string>()->default_value("10.0.0.22:11911"))
			("debug", po::value<unsigned>()->default_value(0))
			("device", po::value<std::string>()->default_value("mlx5_0"))
			("inference", po::value<int>()->default_value(0))
			("precision", po::value<std::string>()->default_value("float")) 
			("help", "Show this help")
			;
    po::variables_map vm;
    po::store(po::parse_command_line(argc, argv, desc), vm);

    if(vm.count("help")) {
    	std::cout << desc;
    	return -1;
    }
    inference = vm["inference"].as<int>();

#if CUDNN_MAJOR >= 6
    std::string precision;
    if (inference)
        precision = "int8";
    else
        precision = "half";
#else
    std::string precision = "float";
#endif
    std::cout << vm.count("precision") << vm["precision"].as<std::string>() << std::endl;
    if (vm.count("precision")) {
        precision = vm["precision"].as<std::string>();
    }

    // Create KVStore
    Component::IKVStore *  store = create_store(vm["dawn-server"].as<std::string>(),
    							vm["device"].as<std::string>(),
								vm["debug"].as<unsigned>());
    // Create pool
    auto pool = store->create_pool("/poolsgpu0", MB(128));


    // Handles to various cuda libraries, structures
    hiprandGenerator_t curand_gen;


    hipFree(0);

    // Initialize curand_gen and set appropriate seed.
    hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curand_gen, 123ULL);


    if (inference) {
        std::cout << std::setw(45) << "Running inference benchmark " << std::endl;
    } else {
        std::cout << std::setw(45) << "Running training benchmark " << std::endl;
    }

    std::cout << std::setw(30) << "Times" << std::endl;
    std::cout << std::setfill('-') << std::setw(190) << "-" << std::endl;
    std::cout << std::setfill(' ');
    std::cout << "   w      h      c      n      k      f_w    f_h  pad_w  pad_h    stride_w  stride_h    precision  fwd_time (usec)  ";

    if (!inference) {
        std::cout << "bwd_inputs_time (usec)  bwd_params_time (usec)  ";
        std::cout << "total_time (usec)";
    }

    if (PAD_KERNELS && ((precision == "int8" && inference) || (USE_TENSOR_CORES && !inference)))
        std::cout << " pad_kerenels  ";

    std::cout << "   fwd_algo " << std::endl;

    std::cout << std::setfill('-') << std::setw(200) << "-" << std::endl;
    std::cout << std::setfill(' ');

    int pad_kernels_count = 0;

    for (const auto &problem : (inference ? inference_server_set : training_set)) {

        // Filter parameters
        int k, c, r, s; // r - filter_h (f_h), s - filter_w (f_w)

        // Input parameters
        int n, w, h;

        // Padding
        int pad_w, pad_h;

        // Stride
        int wstride, hstride;

        std::tie(w, h, c, n, k, s, r, pad_w, pad_h, wstride, hstride) = problem;

        bool skip_kernel = false;
        bool need_padding = false;

#if CUDNN_MAJOR >= 6
        int padded_c, padded_w, padded_h;
        int pad_value;

        padded_c = c;
        padded_h = h;
        padded_w = w;

        if (precision == "int8") {
            pad_value = 4;
            if (c % pad_value || w % pad_value || h % pad_value) {
                pad_kernels_count++;
                if (PAD_KERNELS) {
                    pad_dim(padded_c, pad_value);
                    pad_dim(padded_h, pad_value);
                    pad_dim(padded_w, pad_value);
                    need_padding = true;
                } else {
                    skip_kernel = true;
                }
            }
        }
#if (USE_TENSOR_CORES)
        // Tensor cores need channels to be a multiple of 8. So, added padding for some kernels.
        if (!inference) {
            pad_value = 8;
            if (c % pad_value) {
                pad_kernels_count++;
                if (PAD_KERNELS) {
                    pad_dim(padded_c, pad_value);
                    need_padding = true;
                } else {
                    skip_kernel = true;
                }
            }
        }
#endif
#endif

        int fwd_time, bwd_inputs_time, bwd_params_time;
        std::string fwd_algo_s;

        std::stringstream ss;
        ss << "Unsupported precision requested. Precision: " << precision << " Inference: " << inference;
        //if(w*h*c*n >= MB(8) || padded_w*padded_h*padded_c*n >= MB(8)) {
        //   PLOG(" > 8MB, skip");
        //   continue;
        //} 
#if CUDNN_MAJOR >= 6
        if (precision == "float") {
            std::tie(fwd_time, bwd_inputs_time, bwd_params_time, fwd_algo_s) =
                time_cnn<float, float>(k, padded_c, r, s, n, padded_h, padded_w, pad_h, pad_w, hstride, wstride, num_repeats, curand_gen, inference, store, pool);
        } else if (precision == "half") {
            std::tie(fwd_time, bwd_inputs_time, bwd_params_time, fwd_algo_s) =
                time_cnn<uint16_t, uint16_t>(k, padded_c, r, s, n, padded_h, padded_w, pad_h, pad_w, hstride, wstride, num_repeats, curand_gen, inference, store, pool);
        } else if ((precision == "int8") && inference) {
            if (!skip_kernel) {
                std::tie(fwd_time, bwd_inputs_time, bwd_params_time, fwd_algo_s) =
                    time_cnn<uint8_t, int>(k, padded_c, r, s, n, padded_h, padded_w, pad_h, pad_w, hstride, wstride, num_repeats, curand_gen, inference, store, pool);
            }
        } else {
            throw std::runtime_error(ss.str());
        }
#else
        if (precision != "float")
            throw std::runtime_error(ss.str());
        std::tie(fwd_time, bwd_inputs_time, bwd_params_time, fwd_algo_s) =
            time_cnn<float, float>(k, c, r, s, n, h, w, pad_h, pad_w, hstride, wstride, num_repeats, curand_gen, inference, store, pool);
#endif

        std::cout << std::setw(5) << w;
        std::cout << std::setw(7) << h;
        std::cout << std::setw(7) << c;
        std::cout << std::setw(7) << n;
        std::cout << std::setw(7) << k;
        std::cout << std::setw(7) << s;
        std::cout << std::setw(7) << r;
        std::cout << std::setw(7) << pad_w;
        std::cout << std::setw(8) << pad_h;
        std::cout << std::setw(10) << wstride;
        std::cout << std::setw(10) << hstride;
        std::cout << std::setw(10) << precision;
        std::cout << std::setw(15) << std::setprecision(7);

        if (skip_kernel) {
            std::cout << "Not Supported";
        } else {
            std::cout << fwd_time;
        }

        if (PAD_KERNELS && precision == "int8" && inference) {
            std::cout << std::setw(15) <<  need_padding;
        }



        if (!inference) {
            std::cout << std::setw(24) << std::setprecision(7) << bwd_inputs_time;
            std::cout << std::setw(24) << std::setprecision(7) << bwd_params_time;
            std::cout << std::setw(19) << std::setprecision(8) << fwd_time + bwd_inputs_time + bwd_params_time;
        }

        if (USE_TENSOR_CORES && PAD_KERNELS && !inference) {
            std::cout << std::setw(15) <<  need_padding;
        }


        std::cout << std::setw(25) << fwd_algo_s;
        std::cout << std::endl;
    }

    if (precision == "int8") {
        std::cout << " Total kernels ";
        if (PAD_KERNELS)
            std::cout << "padded: " << pad_kernels_count << std::endl;
        else
            std::cout << "skipped: " << pad_kernels_count << std::endl;

        std::cout << " Total kernels: " << inference_server_set.size() << std::endl;
    }
    // Close the pool
    store->close_pool(pool);
    store->delete_pool("/poolsgpu0");
    // Release the KVStore
    store->release_ref();

    // Destroy all the handles
    hiprandDestroyGenerator(curand_gen);

    return 0;
}
